#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <hipcub/hipcub.hpp>

#ifndef PREFIX
#define PREFIX
#endif

#ifndef LABEL
#define LABEL ""
#endif

#ifndef MATH_FUNC
#define MATH_FUNC tanh
#endif

#ifndef SIZE
#define SIZE 100'000'000
#endif

#ifndef ITERATIONS
#define ITERATIONS 1
#endif


#define FUNC_NAME(x) PREFIX##x

template<class T>
__host__ __device__ T FUNC_NAME(ceil_div)(T dividend, T divisor) {
    return (dividend + divisor-1) / divisor;
}
__global__ void FUNC_NAME(kernel)(float* arr, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N){
    arr[i] = MATH_FUNC(i);
  }
}

__global__ void FUNC_NAME(init_seed)(hiprandState *state, int seed) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init(seed, idx, 0, &state[idx]);
}
__global__ void FUNC_NAME(fillRand)(hiprandState *state, float *arr, int N) {
 int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    arr[i] = hiprand_uniform(&state[i]);
  }
}

extern int reduce_gpu_array()

void FUNC_NAME(main)(int printTime, int correctness) {
  int deviceIdx = 0;
  hipSetDevice(deviceIdx);
  int N = SIZE; // 100M elements
  int block_size = 256;
  float* arr;
  hipMalloc(&arr, N * sizeof(float));
  float grid_size = ceil_div(N, block_size);

  hiprandState *d_state;
  hipMalloc(&d_state, (N* sizeof(hiprandState)));
  FUNC_NAME(init_seed)<<<grid_size, block_size>>>(d_state, time(NULL));
  FUNC_NAME(fillRand)<<<grid_size, block_size>>>(d_state, arr, N);


  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, nullptr);
  FUNC_NAME(kernel)<<<grid_size, block_size>>>(arr, N);
  hipDeviceSynchronize(); // Wait for the GPU to finish
  hipEventRecord(stop, nullptr);
  hipEventSynchronize(start);
  hipEventSynchronize(stop);

  float elapsed;
  hipEventElapsedTime(&elapsed, start, stop);
  if (printTime) printf(LABEL"Time: %f ms\n", elapsed);

  if (correctness) {
    float* sum;
    hipMalloc(&sum, sizeof(float));
    void *temp_storage = nullptr;
    size_t n_temp_storage = 0;
    hipcub::DeviceReduce::Sum(temp_storage, n_temp_storage, arr, sum, N);
    hipMalloc(&temp_storage, n_temp_storage);
    hipcub::DeviceReduce::Sum(temp_storage, n_temp_storage, arr, sum, N);
    float sum_host;
    hipMemcpy(&sum_host, sum, sizeof(float), hipMemcpyDeviceToHost);
    printf(LABEL"Sum: %f\n", sum_host);
  }
}
