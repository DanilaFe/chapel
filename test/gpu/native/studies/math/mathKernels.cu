#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <hipcub/hipcub.hpp>
#include <math.h>

#ifndef mk_PREFIX
#define mk_PREFIX
#endif

#ifndef mk_LABEL
#define mk_LABEL ""
#endif

#ifndef mk_MATH_FUNC
#define mk_MATH_FUNC tanh
#endif

#ifndef mk_SIZE
#define mk_SIZE 100'000'000
#endif

#ifndef mk_ITERATIONS
#define mk_ITERATIONS 1
#endif


#define mk_FUNC_NAME_inner2(a, b) a ## b
#define mk_FUNC_NAME_inner1(a, b) mk_FUNC_NAME_inner2(a, b)
#define mk_FUNC_NAME(a) mk_FUNC_NAME_inner1(mk_PREFIX, a)

int mk_FUNC_NAME(ceil_div)(int dividend, int divisor) {
    return (dividend + divisor-1) / divisor;
}
__global__ void mk_FUNC_NAME(kernel)(float* arr, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N){
    arr[i] = mk_MATH_FUNC(i);
  }
}

__global__ void mk_FUNC_NAME(init_seed)(hiprandState *state, int seed) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init(seed, idx, 0, &state[idx]);
}
__global__ void mk_FUNC_NAME(fillRand)(hiprandState *state, float *arr, int N) {
 int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    arr[i] = hiprand_uniform(&state[i]);
  }
}

// extern "C"
void mk_FUNC_NAME(main)(int printTime, int correctness) {
  int deviceIdx = 0;
  hipSetDevice(deviceIdx);
  int N = mk_SIZE; // 100M elements
  int block_size = 256;
  float* arr;
  hipMalloc(&arr, N * sizeof(float));
  float grid_size = mk_FUNC_NAME(ceil_div)(N, block_size);

  hiprandState *d_state;
  hipMalloc(&d_state, (N* sizeof(hiprandState)));
  mk_FUNC_NAME(init_seed)<<<grid_size, block_size>>>(d_state, time(NULL));
  mk_FUNC_NAME(fillRand)<<<grid_size, block_size>>>(d_state, arr, N);


  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, nullptr);
  mk_FUNC_NAME(kernel)<<<grid_size, block_size>>>(arr, N);
  hipDeviceSynchronize(); // Wait for the GPU to finish
  hipEventRecord(stop, nullptr);
  hipEventSynchronize(start);
  hipEventSynchronize(stop);

  float elapsed;
  hipEventElapsedTime(&elapsed, start, stop);
  if (printTime)
    printf(mk_LABEL"Time: %f ms\n", elapsed);

  if (correctness) {
    float* sum;
    hipMalloc(&sum, sizeof(float));
    void *temp_storage = nullptr;
    size_t n_temp_storage = 0;
    hipcub::DeviceReduce::Sum(temp_storage, n_temp_storage, arr, sum, N);
    hipMalloc(&temp_storage, n_temp_storage);
    hipcub::DeviceReduce::Sum(temp_storage, n_temp_storage, arr, sum, N);
    float sum_host;
    hipMemcpy(&sum_host, sum, sizeof(float), hipMemcpyDeviceToHost);
    printf(mk_LABEL"Sum: %f\n", sum_host);
  }
}
